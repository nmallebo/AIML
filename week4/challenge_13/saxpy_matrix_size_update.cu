

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    for (int power = 15; power <= 25; power++) {
        int N = 1 << power;
        printf("N = 2^%d = %d elements\n", power, N);

        float *x, *y, *d_x, *d_y;
        x = (float*)malloc(N * sizeof(float));
        y = (float*)malloc(N * sizeof(float));

        hipMalloc(&d_x, N * sizeof(float)); 
        hipMalloc(&d_y, N * sizeof(float));

        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

        // CUDA event objects for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Start recording
        hipEventRecord(start);

        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

        // Stop recording
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

        float maxError = 0.0f;
        for (int i = 0; i < N; i++)
            maxError = fmax(maxError, fabs(y[i] - 4.0f));

        printf("Max error: %f\n", maxError);
        printf("Execution time: %f ms\n", milliseconds);
        printf("Throughput: %f GFLOPS\n", (2.0 * N / (milliseconds / 1000.0)) / 1e9);
        printf("--------------------------------------\n");

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);
    }

    return 0;
}

