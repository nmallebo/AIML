#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    FILE *fp = fopen("timing_results.csv", "w");
    if (fp == NULL) {
        printf("Failed to open file for writing!\n");
        return -1;
    }
    // CSV Header
    fprintf(fp, "Power,N,HostMallocTime_ms,CudaMallocTime_ms,MemcpyH2DTime_ms,KernelTime_ms,MemcpyD2HTime_ms,HostFreeTime_ms\n");

    for (int power = 15; power <= 25; power++) {
        int N = 1 << power;
        printf("N = 2^%d = %d elements\n", power, N);

        float *x, *y, *d_x, *d_y;

        clock_t host_start = clock();
        x = (float*)malloc(N * sizeof(float));
        y = (float*)malloc(N * sizeof(float));
        clock_t host_end = clock();
        float mallocTime = 1000.0 * (host_end - host_start) / CLOCKS_PER_SEC;

        // CUDA events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // cudaMalloc timing
        hipEventRecord(start);
        hipMalloc(&d_x, N * sizeof(float)); 
        hipMalloc(&d_y, N * sizeof(float));
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float cudaMallocTime = 0;
        hipEventElapsedTime(&cudaMallocTime, start, stop);

        // Initialize host arrays
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        // memcpy H2D timing
        hipEventRecord(start);
        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float memcpyH2DTime = 0;
        hipEventElapsedTime(&memcpyH2DTime, start, stop);

        // Kernel timing
        hipEventRecord(start);
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float kernelTime = 0;
        hipEventElapsedTime(&kernelTime, start, stop);

        // memcpy D2H timing
        hipEventRecord(start);
        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float memcpyD2HTime = 0;
        hipEventElapsedTime(&memcpyD2HTime, start, stop);

        // Check results
        float maxError = 0.0f;
        for (int i = 0; i < N; i++)
            maxError = fmax(maxError, fabs(y[i] - 4.0f));

        printf("Max error: %f\n", maxError);
        printf("Kernel Execution time: %f ms\n", kernelTime);

        // Free timing
        host_start = clock();
        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);
        host_end = clock();
        float freeTime = 1000.0 * (host_end - host_start) / CLOCKS_PER_SEC;

        // Write timings to CSV
        fprintf(fp, "%d,%d,%.4f,%.4f,%.4f,%.4f,%.4f,%.4f\n",
                power, N,
                mallocTime,
                cudaMallocTime,
                memcpyH2DTime,
                kernelTime,
                memcpyD2HTime,
                freeTime);

        hipEventDestroy(start);
        hipEventDestroy(stop);

        printf("--------------------------------------\n");
    }

    fclose(fp);

    return 0;
}
